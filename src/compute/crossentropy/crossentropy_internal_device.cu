#include "hip/hip_runtime.h"

#include "compute/crossentropy/crossentropy_internal.h"

namespace magmadnn {
namespace internal {

template <typename T>
__global__ void kernel_crossentropy_full_device(T *x, T *y, T *softmax, T *out) {}

template <typename T>
void crossentropy_full_device(Tensor<T> *x, Tensor<T> *y, Tensor<T> *softmax, Tensor<T> *out) {}
template void crossentropy_full_device(Tensor<int> *x, Tensor<int> *y, Tensor<int> *softmax, Tensor<int> *out);
template void crossentropy_full_device(Tensor<float> *x, Tensor<float> *y, Tensor<float> *softmax, Tensor<float> *out);
template void crossentropy_full_device(Tensor<double> *x, Tensor<double> *y, Tensor<double> *softmax,
                                       Tensor<double> *out);

}  // namespace internal
}  // namespace magmadnn