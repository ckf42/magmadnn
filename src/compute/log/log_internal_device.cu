#include "hip/hip_runtime.h"
#include "compute/log/log_internal.h"

#define BLK_SIZE 1024

namespace magmadnn {
namespace internal {

template <typename T>
__global__ void kernel_log_full_device(T *x, T *out, unsigned int size, T epsilon) {
    unsigned int idx = blockDim.x * blockIdx.x + threadIdx.x;
    unsigned int stride = blockDim.x * gridDim.x;

    for (unsigned int i = idx; i < size; i += stride) {
        out[i] = log(x[i] + epsilon);
    }
}
template <>
__global__ void kernel_log_full_device(int *x, int *out, unsigned int size, int epsilon) {
    unsigned int idx = blockDim.x * blockIdx.x + threadIdx.x;
    unsigned int stride = blockDim.x * gridDim.x;

    for (unsigned int i = idx; i < size; i += stride) {
        out[i] = (int) log((float) x[i] + epsilon);
    }
}

template <typename T>
void log_full_device(Tensor<T> *x, Tensor<T> *out, bool stable) {
    unsigned int size = x->get_size();
    kernel_log_full_device<<<(size + BLK_SIZE - 1) / BLK_SIZE, BLK_SIZE>>>(x->get_ptr(), out->get_ptr(), size, (stable)?static_cast<T>(1E-8):static_cast<T>(0));
}
template void log_full_device(Tensor<int> *x, Tensor<int> *out, bool stable);
template void log_full_device(Tensor<float> *x, Tensor<float> *out, bool stable);
template void log_full_device(Tensor<double> *x, Tensor<double> *out, bool stable);

template <typename T>
__global__ void kernel_log_grad_device(T *x, T *grad, T *out, unsigned int size, T epsilon) {
    unsigned int idx = blockDim.x * blockIdx.x + threadIdx.x;
    unsigned int stride = blockDim.x * gridDim.x;

    for (unsigned int i = idx; i < size; i += stride) {
        out[i] = grad[i] / (x[i] + epsilon);
    }
}

template <typename T>
void log_grad_device(Tensor<T> *x, Tensor<T> *grad, Tensor<T> *out, bool stable) {
    unsigned int size = x->get_size();
    kernel_log_grad_device<<<(size + BLK_SIZE - 1) / BLK_SIZE, BLK_SIZE>>>(x->get_ptr(), grad->get_ptr(),
                                                                           out->get_ptr(), size, (stable)?static_cast<T>(1E-8):static_cast<T>(0));
}
template void log_grad_device(Tensor<int> *x, Tensor<int> *grad, Tensor<int> *out, bool stable);
template void log_grad_device(Tensor<float> *x, Tensor<float> *grad, Tensor<float> *out, bool stable);
template void log_grad_device(Tensor<double> *x, Tensor<double> *grad, Tensor<double> *out, bool stable);

}  // namespace internal
}  // namespace magmadnn

#undef BLK_SIZE
