#include "hip/hip_runtime.h"
/**
 * @file product_internal_device.cu
 * @author Daniel Nichols
 * @version 0.1
 * @date 2019-05-21
 *
 * @copyright Copyright (c) 2019
 */
#include "compute/product/product_internal.h"

#define BLK_SIZE 1024

namespace magmadnn {
namespace internal {

template <typename T>
__global__ void kernel_product_full_device(T alpha, T *a, T *b, T *out, unsigned int arr_size) {
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int stride = blockDim.x * gridDim.x;

    for (unsigned int i = idx; i < arr_size; i += stride) {
        out[i] = a[i] * b[i];
    }
}

template <typename T>
void product_full_device(T alpha, Tensor<T> *a, Tensor<T> *b, Tensor<T> *out) {
    unsigned int size = out->get_size();
    kernel_product_full_device<<<(size + BLK_SIZE - 1) / BLK_SIZE, BLK_SIZE>>>(alpha, a->get_ptr(), b->get_ptr(),
                                                                               out->get_ptr(), size);
}
template void product_full_device(int alpha, Tensor<int> *a, Tensor<int> *b, Tensor<int> *out);
template void product_full_device(float alpha, Tensor<float> *a, Tensor<float> *b, Tensor<float> *out);
template void product_full_device(double alpha, Tensor<double> *a, Tensor<double> *b, Tensor<double> *out);

template <typename T>
__global__ void kernel_scalar_tensor_product_full_device(T scalar, T *a, T *out, unsigned int arr_size) {
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int stride = blockDim.x * gridDim.x;

    for (unsigned int i = idx; i < arr_size; i += stride) {
        out[i] = scalar * a[i];
    }
}

template <typename T>
void scalar_tensor_product_full_device(T scalar, Tensor<T> *a, Tensor<T> *out) {
    unsigned int size = out->get_size();
    kernel_scalar_tensor_product_full_device<<<(size + BLK_SIZE - 1) / BLK_SIZE, BLK_SIZE>>>(scalar, a->get_ptr(),
                                                                                             out->get_ptr(), size);
}
template void scalar_tensor_product_full_device(int scalar, Tensor<int> *a, Tensor<int> *out);
template void scalar_tensor_product_full_device(float scalar, Tensor<float> *a, Tensor<float> *out);
template void scalar_tensor_product_full_device(double scalar, Tensor<double> *a, Tensor<double> *out);

}  // namespace internal
}  // namespace magmadnn

#undef BLK_SIZE
