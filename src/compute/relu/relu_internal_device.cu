#include "hip/hip_runtime.h"
/**
 * @file relu_internal_device.cu
 * @author Daniel Nichols
 * @version 0.1
 * @date 2019-05-01
 *
 * @copyright Copyright (c) 2019
 */
#include "compute/relu/relu_internal.h"

namespace magmadnn {
namespace internal {

template <typename T>
__global__ void kernel_relu_full_device(unsigned int size, T *arr, T *out) {
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int stride = blockDim.x * gridDim.x;

    for (unsigned int i = idx; i < size; i += stride) {
        if (arr[i] < 0)
            out[i] = 0;
        else
            out[i] = arr[i];
    }
}

template <typename T>
void relu_full_device(Tensor<T> *x, Tensor<T> *out) {
    kernel_relu_full_device<<<x->get_size(), 1>>>(x->get_size(), x->get_ptr(), out->get_ptr());
}
template void relu_full_device(Tensor<int> *x, Tensor<int> *out);
template void relu_full_device(Tensor<float> *x, Tensor<float> *out);
template void relu_full_device(Tensor<double> *x, Tensor<double> *out);

}  // namespace internal
}  // namespace magmadnn