#include "hip/hip_runtime.h"
/**
 * @file sigmoid_internal_device.cu
 * @author Daniel Nichols
 * @version 1.0
 * @date 2019-02-23
 *
 * @copyright Copyright (c) 2019
 */
#include "compute/sigmoid/sigmoid_internal.h"

#define BLK_SIZE 1024

namespace magmadnn {
namespace internal {

template <typename T>
__global__ void kernel_fast_sigmoid_full_device(unsigned int size, T *x, T *out) {
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int stride = blockDim.x * gridDim.x;

    for (unsigned int i = idx; i < size; i += stride) {
        out[i] = x[i] / (1 + abs(x[i]));
    }
}

template <typename T>
__global__ void kernel_sigmoid_full_device(unsigned int size, T *x, T *out) {
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int stride = blockDim.x * gridDim.x;

    for (unsigned int i = idx; i < size; i += stride) {
        out[i] = 1 / (1 + exp(-x[i]));
    }
}

/* exp(INT_TYPE) is not defined in CUDA, so just use 1/(1+|x|) for int.
   Everything will be zero anyways. TODO: decide what to do with int sigmoid. */
template <>
__global__ void kernel_sigmoid_full_device(unsigned int size, int *x, int *out) {
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int stride = blockDim.x * gridDim.x;

    for (unsigned int i = idx; i < size; i += stride) {
        out[i] = 1 / (1 + abs(x[i]));
    }
}

template <typename T>
void sigmoid_full_device(Tensor<T> *x, Tensor<T> *out, bool fast) {
    unsigned int size = out->get_size();

    if (fast)
        kernel_fast_sigmoid_full_device<<<(size + BLK_SIZE - 1) / BLK_SIZE, BLK_SIZE>>>(size, x->get_ptr(),
                                                                                        out->get_ptr());
    else
        kernel_sigmoid_full_device<<<(size + BLK_SIZE - 1) / BLK_SIZE, BLK_SIZE>>>(size, x->get_ptr(), out->get_ptr());
}

template <>
void sigmoid_full_device(Tensor<int> *x, Tensor<int> *out, bool fast) {
    /* sigmoid doesn't make much sense on integer precision */
    for (unsigned int i = 0; i < x->get_size(); i++) x->set(i, (int) exp(x->get(i)));
}

template void sigmoid_full_device(Tensor<float> *x, Tensor<float> *out, bool fast);
template void sigmoid_full_device(Tensor<double> *x, Tensor<double> *out, bool fast);

template <typename T>
__global__ void kernel_sigmoid_grad_device(T *output, T *grad, T *out, unsigned int size, bool is_grad_scalar) {
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int stride = blockDim.x * gridDim.x;

    for (unsigned int i = idx; i < size; i += stride) {
        out[i] = grad[(is_grad_scalar) ? 0 : i] * output[i] * (1 - output[i]);
    }
}

template <typename T>
void sigmoid_grad_device(Tensor<T> *output, Tensor<T> *grad, Tensor<T> *out) {
    unsigned int size = out->get_size();

    kernel_sigmoid_grad_device<<<(size + BLK_SIZE - 1) / BLK_SIZE, BLK_SIZE>>>(
        output->get_ptr(), grad->get_ptr(), out->get_ptr(), size, (grad->get_size() == 1));
}
template void sigmoid_grad_device(Tensor<int> *output, Tensor<int> *grad, Tensor<int> *out);
template void sigmoid_grad_device(Tensor<float> *output, Tensor<float> *grad, Tensor<float> *out);
template void sigmoid_grad_device(Tensor<double> *output, Tensor<double> *grad, Tensor<double> *out);

}  // namespace internal
}  // namespace magmadnn

#undef BLK_SIZE