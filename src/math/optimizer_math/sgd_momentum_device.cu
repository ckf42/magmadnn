#include "hip/hip_runtime.h"
/**
 * @file sgd_momentum_device.cu
 * @author Sedrick Keh
 * @version 1.0
 * @date 2019-07-26
 *
 * @copyright Copyright (c) 2019
 */
 #include "math/optimizer_math/sgd_momentum.h"

 #define BLK_SIZE 1024
 
 namespace magmadnn {
 namespace math {
 
 template <typename T>
 __global__ void kernel_sgd_momentum_device(T learning_rate, T momentum, T *prev, T* grad, T *out, unsigned int size) {
     unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
     unsigned int stride = blockDim.x * gridDim.x;
 
     for (unsigned int i = idx; i < size; i += stride) {
         prev[i] = momentum * prev[i] + (1 - momentum) * grad[i];
         out[i] = out[i] - learning_rate * prev[i];
     }
 }
 
 template <typename T>
 void sgd_momentum_device(T learning_rate, T momentum, Tensor<T> *prev, Tensor<T> *grad, Tensor<T> *out) {
     unsigned int size = out->get_size();
     kernel_sgd_momentum_device<<<(size + BLK_SIZE - 1) / BLK_SIZE, BLK_SIZE>>>(learning_rate, momentum, prev->get_ptr(), 
                                                                                grad->get_ptr(), out->get_ptr(), size);
 }
 template void sgd_momentum_device(int learning_rate, int momentum, Tensor<int> *prev, Tensor<int> *grad, Tensor<int> *out);
 template void sgd_momentum_device(float learning_rate, float momentum, Tensor<float> *prev, Tensor<float> *grad, Tensor<float> *out);
 template void sgd_momentum_device(double learning_rate, double momentum, Tensor<double> *prev, Tensor<double> *grad, Tensor<double> *out);
 
 }  // namespace math
 }  // namespace magmadnn
 
 #undef BLK_SIZE